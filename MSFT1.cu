#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>  // Include the CURAND header

#include"FourierMethods.h"
#include"FFT.h"



void FFT_CGMY_PDF(double* x, double* f, int N_points, int N_FFT_TERMS ,double u_max, double T, double S0, double r, double q, double C, double G, double M, double Y)
{
	double x_min = x[0];
	double du = u_max / N_FFT_TERMS;
	double dx = 2 * pi / (N_FFT_TERMS * du);

	Complex_v F(N_FFT_TERMS);
	for (int k = 0; k < N_FFT_TERMS; k++)
	{
		double u = du * k;
		F[k] = CF_CGMY(u, T, S0, r, q, C, G, M, Y);
	}


	Complex_v phi(N_FFT_TERMS), phi_boundary(N_FFT_TERMS);
	for (int k = 0; k < N_FFT_TERMS; ++k) {
		double x = x_min + dx * k;
		double u = du * k;
		phi[k] = exp(-i * x_min * u) * F[k];
		Complex gamma1 = F[0];
		Complex gamma2 = exp(-i * x * u_max) * F[N_FFT_TERMS - 1];
		phi_boundary[k] = 0.5 * (gamma1 + gamma2);
	}

	FFT(phi);
	LinearInterpolation li;
	Double_v f_adj(N_points), x_adj(N_points);
	for (int k = 0; k < N_points; k++)
	{
		double x_k = x_min + dx * k;
		double f_k = real(phi[k] - phi_boundary[k]) * du / pi;
		f_adj[k] = f_k;
		x_adj[k] = x_k;
		li.AddPoint(x_k, f_k);
	}

	//tk::spline s(x_adj, f_adj);	//spline interpolation

	//Double_v f(N_points);
	for (int k = 0; k < N_points; k++)
	{
		f[k] = li.value(x[k]);
		//f[k] = s(x[k]);
	}
}



void COS_CGMY_pdf(double* x, double* f, int N_points, int N, double T, double S0, double r, double q, double C, double G, double M, double Y, double a, double b)
{
	Double_v F;
	for (int k = 0; k < N; k++)
	{
		double w = -C * tgamma(-Y) * (pow(M - 1, Y) - pow(M, Y) + pow(G + 1, Y) - pow(G, Y));
		double mu = r + w - q;
		double u_k = k * pi / (b - a);
		double F_k = 2 / (b - a) * real(CF_CGMY(u_k, T, S0, r, q, C, G, M, Y) /** exp(-i * u_k * (log(S0) + mu * T))*/ * exp(-i * u_k * a));
		F.push_back(F_k);
	}
	F[0] *= 0.5;

	for (int k = 0; k < N_points; k++)
	{
		double x_k = x[k];
		double f_x = 0;
		for (int k = 0; k < N; k++) {
			double u_k = k * pi / (b - a);
			double costerm = cos(u_k * (x_k - a));
			f_x += F[k] * costerm;
		}
		f[k] = f_x;
	}
}


// CUDA kernel function to price CGMY option
__global__ void PriceByMC(float* price, double S0, double r, double q, double w, double T, double K, int N_sim, double* ST) {
	unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int totalThreads = blockDim.x * gridDim.x;

	float localPrice = 0;

	for (int i = idx; i < N_sim; i += totalThreads)
	{
		double S = S0 * exp((r - q + w) * T + ST[i]);
		double payoff = max(K - S, 0.0);
		localPrice += payoff * exp(-r * T) / N_sim;
	}

	atomicAdd(price, localPrice);
}


// Linear interpolation kernel
__global__ void linearInterpolateKernel(double* d_x, double* d_y, double* d_interp_x, double* d_interp_y, int J, int N_sim) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < N_sim) {
		double x = d_interp_x[idx];

		// Find the interval x is in
		for (int i = 0; i < J - 1; i++) {
			//in case of out of boundaries assign closest value
			if (x < d_x[i]) { d_interp_y[idx] = d_y[i]; break; }
			if (x > d_x[J - 1]) { d_interp_y[idx] = d_y[J - 1]; break; }
			if (x >= d_x[i] && x <= d_x[i + 1]) {
				double x0 = d_x[i];
				double x1 = d_x[i + 1];
				double y0 = d_y[i];
				double y1 = d_y[i + 1];

				// Perform linear interpolation
				d_interp_y[idx] = y0 + (x - x0) / (x1 - x0) * (y1 - y0);
				break;
			}
		}
	}
}


//Give a randState to each CUDA thread from which it can sample from
__global__ void init_rng(unsigned int seed, hiprandState* state)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, idx, 0, &state[idx]);
}

__global__ void gen_x(hiprandState* state, double* x)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	hiprandState localState = state[idx];
	x[idx] = hiprand_uniform_double(&localState);
	state[idx] = localState;
}





double CGMY_COS_CUDA(double S0, double  K, double  T, double  r, double  C, double  G, double  M, double  Y, double q, double D, int N_sim) {

	////////////////////////////////CDF COMPUTATION////////////////////////////////

	// Number of points to generate and CUDA parameters
	const int threadsPerBlock = 1024;
	const int numBlocks = (N_sim + threadsPerBlock - 1) / threadsPerBlock;

	//COS parameters
	double L = 10;
	const int N_COS_TERMS = int(pow(2, 14));
	double a = -L * sqrt(T);
	double b = L * sqrt(T);

	//size of a pdf/cdf
	int N_points = 1000; //number of points to recover
	double x_min = -D;
	double x_max = D;
	double dx = (x_max - x_min) / N_points;


	double* h_x = new double[N_points];
	for (int j = 0; j < N_points; j++) h_x[j] = x_min + j * dx;

	double* f_x = new double[N_points];

	//COS_CGMY_pdf(h_x, f_x, N_points, N_COS_TERMS, T, S0, r, q, C, G, M, Y, a, b);
	FFT_CGMY_PDF(h_x, f_x, N_points, N_COS_TERMS , 20, T, S0, r, q, C, G, M, Y);

	int J_size = 0;
	double F_j = f_x[0];
	for (int j = 0; j < N_points; j++)
	{
		F_j += (j == 0) ? 0 : f_x[j] * dx;
		if (0.05 <= F_j && F_j <= 0.95) J_size += 1;
		//cout << h_x[j] << '\t' << F_j << endl;
	}

	double* F_x;
	hipMallocManaged((void**)&F_x, J_size * sizeof(double));
	double* x;
	hipMallocManaged((void**)&x, J_size * sizeof(double));

	F_j = f_x[0]; int k = 0;
	for (int j = 0; j < N_points; j++)
	{
		F_j += (j == 0) ? 0 : f_x[j] * dx;
		if (0.05 <= F_j && F_j <= 0.95)
		{
			F_x[k] = F_j;
			x[k] = h_x[j];
			k++;
		}
	}

	delete[] f_x;
	delete[] h_x;

	////////////////////////////////SAMPLING OF UNIFORMAL VARIABLES////////////////////////////////

	double* d_uniform;
	hipMalloc((void**)&d_uniform, N_sim * sizeof(double));

	hiprandState* d_state;
	hipMalloc((void**)&d_state, N_sim * sizeof(hiprandState));

	unsigned long seed = time(NULL);
	// Init the random states
	init_rng << <numBlocks, threadsPerBlock >> > (seed, d_state);
	// Generate numbers in GPU
	gen_x << <numBlocks, threadsPerBlock >> > (d_state, d_uniform);

	////////////////////////////////INTERPOLATING WITH CDF////////////////////////////////

	double* d_ST;
	hipMallocManaged((void**)&d_ST, N_sim * sizeof(double));

	linearInterpolateKernel << <numBlocks, threadsPerBlock >> > (F_x, x, d_uniform, d_ST, J_size, N_sim);

	hipFree(d_uniform);
	hipFree(F_x);
	hipFree(x);

	////////////////////////////////PRICING OPTIONS////////////////////////////////

	float h_price;
	float* d_price;

	hipMalloc((void**)&d_price, sizeof(float));
	hipMemset(d_price, 0, sizeof(float));

	double w = -C * tgamma(-Y) * (pow(M - 1, Y) - pow(M, Y) + pow(G + 1, Y) - pow(G, Y));

	PriceByMC << <numBlocks, threadsPerBlock >> > (d_price, S0, r, q, w, T, K, N_sim, d_ST);

	hipMemcpy(&h_price, d_price, sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_price);

	hipDeviceReset();

	return double(h_price);
}

