#include "hip/hip_runtime.h"
﻿
#include"FourierMethods.h"


Complex CF_CGMY(Complex u, double T, double S0, double r, double q, double C, double G, double M, double Y)
{
	double w = -C * tgamma(-Y) * (pow(M - 1, Y) - pow(M, Y) + pow(G + 1, Y) - pow(G, Y));
	double mu = r + w - q;
	Complex phi_CGMY = exp(C * T * tgamma(-Y) * (pow(M - i * u, Y) - pow(M, Y) + pow(G + i * u, Y) - pow(G, Y)));
	return exp(i * u * (log(S0) + mu * T)) * phi_CGMY;
}

void COS_CGMY_pdf(double* x, double* f, int J, int N, double T, double S0, double r, double q, double C, double G, double M, double Y, double a, double b)
{
	Double_v F;
	for (int k = 0; k < N; k++)
	{
		double w = -C * tgamma(-Y) * (pow(M - 1, Y) - pow(M, Y) + pow(G + 1, Y) - pow(G, Y));
		double mu = r + w - q;
		double u_k = k * pi / (b - a);
		double F_k = 2 / (b - a) * real(CF_CGMY(u_k, T, S0, r, q, C, G, M, Y) * exp(-i * u_k * (log(S0) + mu * T)) * exp(-i * u_k * a));
		F.push_back(F_k);
	}
	F[0] *= 0.5;

	for (int k = 0; k < J; k++)
	{
		double x_k = x[k];
		double f_x = 0;
		for (int k = 0; k < N; k++) {
			double u_k = k * pi / (b - a);
			double costerm = cos(u_k * (x_k - a));
			f_x += F[k] * costerm;
		}
		f[k] = f_x;
	}
}


// CUDA kernel function to price CGMY option
__global__ void PriceByMC(float* price, double S0, double r, double q, double w, double T, double K, int N_sim, double* ST) {
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int totalThreads = blockDim.x * gridDim.x;

	float localPrice = 0;

	for (int i = tid; i < N_sim; i += totalThreads)
	{
		double S = S0 * exp((r - q + w) * T + ST[i]);
		double payoff = max(K - S, 0.0);
		//localPrice.add(payoff * exp(-r * T));
		localPrice += payoff * exp(-r * T) / N_sim;
	}

	atomicAdd(price, localPrice);
}


// Linear interpolation kernel
__global__ void linearInterpolateKernel(double* d_x, double* d_y, double* d_interp_x, double* d_interp_y, int J, int N_sim) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < N_sim) {
		double x = d_interp_x[idx];

		// Find the interval x is in
		for (int i = 0; i < J - 1; i++) {
			if (x >= d_x[i] && x <= d_x[i + 1]) {
				double x0 = d_x[i];
				double x1 = d_x[i + 1];
				double y0 = d_y[i];
				double y1 = d_y[i + 1];

				// Perform linear interpolation
				d_interp_y[idx] = y0 + (x - x0) / (x1 - x0) * (y1 - y0);
				break;
			}
		}
	}
}


//Give a randState to each CUDA thread from which it can sample from
__global__ void init_rng(unsigned int seed, hiprandState* state)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, idx, 0, &state[idx]);
}

__global__ void gen_x(hiprandState* state, double* x)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	hiprandState localState = state[idx];
	x[idx] = hiprand_uniform_double(&localState);
	state[idx] = localState;
}





void CGMY_MC() {

	////////////////////////////////CDF COMPUTATION////////////////////////////////
	double S0(10), K(10), T(0.25), r(0.1), C(0.22), G(0.75), M(1.0), Y(1.8), q(0.0);

	// Number of points to generate and CUDA parameters
	const int N_sim = 1e7;
	const int threadsPerBlock = 1024;
	const int numBlocks = (N_sim + threadsPerBlock - 1) / threadsPerBlock;

	//COS parameters
	double L = 8;
	const int N_COS_TERMS = int(pow(2, 14));
	double a = -L * sqrt(T);
	double b = L * sqrt(T);

	//size of a pdf/cdf
	int J = 1000;
	double x_min = -4;
	double x_max = 4;
	double dx = (x_max - x_min) / J;


	double* x;
	hipMallocManaged((void**)&x, J * sizeof(double));
	for (int j = 0; j < J; j++) x[j] = x_min + j * dx;

	double* f_x = new double[J];

	COS_CGMY_pdf(x, f_x, J, N_COS_TERMS, T, S0, r, q, C, G, M, Y, a, b);

	double* F_x;
	hipMallocManaged((void**)&F_x, J * sizeof(double));

	double F_j = f_x[0];
	for (int j = 0; j < J; j++)
	{
		F_j += (j == 0) ? 0 : f_x[j];
		F_x[j] = F_j * dx;
	}


	delete[] f_x;

	double w = -C * tgamma(-Y) * (pow(M - 1, Y) - pow(M, Y) + pow(G + 1, Y) - pow(G, Y));
	double mu = r + w - q;


	////////////////////////////////SAMPLING OF UNIFORMAL VARIABLES////////////////////////////////
	auto start = high_resolution_clock::now();
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);


	double* d_uniform;
	//Allocate Unified Memory – accessible from CPU or GPU
	hipMallocManaged((void**)&d_uniform, N_sim * sizeof(double));

	// Create a device pointer on the host, to hold the random states
	hiprandState* d_state;
	hipMalloc((void**)&d_state, N_sim * sizeof(hiprandState));

	unsigned long seed = time(NULL);
	// Init the random states
	init_rng << <numBlocks, threadsPerBlock >> > (seed, d_state);
	// Generate numbers in GPU
	gen_x << <numBlocks, threadsPerBlock >> > (d_state, d_uniform);

	////////////////////////////////INTERPOLATING WITH CDF////////////////////////////////

	double* d_ST;
	hipMallocManaged((void**)&d_ST, N_sim * sizeof(double));

	linearInterpolateKernel << <numBlocks, threadsPerBlock >> > (F_x, x, d_uniform, d_ST, J, N_sim);

	hipFree(d_uniform);
	hipFree(F_x);
	hipFree(x);

	////////////////////////////////PRICING OPTIONS////////////////////////////////

	float h_price;
	float* d_price;

	hipMalloc((void**)&d_price, sizeof(float));
	hipMemset(d_price, 0, sizeof(float));

	PriceByMC << <numBlocks, threadsPerBlock >> > (d_price, S0, r, q, w, T, K, N_sim, d_ST);

	hipMemcpy(&h_price, d_price, sizeof(float), hipMemcpyDeviceToHost);
	stop = high_resolution_clock::now(); duration = duration_cast<microseconds>(stop - start);
	hipFree(d_price);

	cout << "GPU price is " << h_price << endl;
	cout << "GPU pricing time: " << duration.count() / 1000 << " milliseconds" << endl;


	start = high_resolution_clock::now();
	CGMYmarket CGMYoption1(S0, r, q, C, G, M, Y);
	double cpuPrice = CGMYoption1.PriceByMonteCarlo(K, T, N_sim, false);
	cout << "CPU price is " << cpuPrice << endl;
	stop = high_resolution_clock::now(); duration = duration_cast<microseconds>(stop - start);
	cout << "CPU pricing time: " << duration.count() / 1000 << " milliseconds" << endl;


	cout << endl << CGMYoption1.PriceByCarrMadanDirect(K, T, 1.5, 2048, 1e-2, false) << endl;
	cout << endl << CGMYoption1.PriceByFST(K, T, 4096, false) << endl;
	cout << endl << CGMYoption1.PriceByCOS(K, T, 2048, false) << endl;
}

