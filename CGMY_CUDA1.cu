#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>  // Include the CURAND header

#include"FourierMethods.h"


Complex CF_CGMY(Complex u, double T, double S0, double r, double q, double C, double G, double M, double Y)
{
	double w = -C * tgamma(-Y) * (pow(M - 1, Y) - pow(M, Y) + pow(G + 1, Y) - pow(G, Y));
	double mu = r + w - q;
	Complex phi_CGMY = exp(C * T * tgamma(-Y) * (pow(M - i * u, Y) - pow(M, Y) + pow(G + i * u, Y) - pow(G, Y)));
	return exp(i * u * (log(S0) + mu * T)) * phi_CGMY;
}

void COS_CGMY_pdf(double* x, double* f, int J, int N, double T, double S0, double r, double q, double C, double G, double M, double Y, double a, double b)
{
	Double_v F;
	for (int k = 0; k < N; k++)
	{
		double w = -C * tgamma(-Y) * (pow(M - 1, Y) - pow(M, Y) + pow(G + 1, Y) - pow(G, Y));
		double mu = r + w - q;
		double u_k = k * pi / (b - a);
		double F_k = 2 / (b - a) * real(CF_CGMY(u_k, T, S0, r, q, C, G, M, Y) * exp(-i * u_k * (log(S0) + mu * T)) * exp(-i * u_k * a));
		F.push_back(F_k);
	}
	F[0] *= 0.5;

	for (int k = 0; k < J; k++)
	{
		double x_k = x[k];
		double f_x = 0;
		for (int k = 0; k < N; k++) {
			double u_k = k * pi / (b - a);
			double costerm = cos(u_k * (x_k - a));
			f_x += F[k] * costerm;
		}
		f[k] = f_x;
	}
}


// CUDA kernel function to price CGMY option
__global__ void PriceByMC(float* price, double S0, double r, double q, double w, double T, double K, int N_sim, double* ST) {
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
	unsigned int totalThreads = blockDim.x * gridDim.x;

	float localPrice = 0;

	for (int i = tid; i < N_sim; i += totalThreads)
	{
		double S = S0 * exp((r - q + w) * T + ST[i]);
		double payoff = max(K - S, 0.0);
		localPrice += payoff * exp(-r * T) / N_sim;
	}

	atomicAdd(price, localPrice);
}


// Linear interpolation kernel
__global__ void linearInterpolateKernel(double* d_x, double* d_y, double* d_interp_x, double* d_interp_y, int J, int N_sim) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < N_sim) {
		double x = d_interp_x[idx];

		// Find the interval x is in
		for (int i = 0; i < J - 1; i++) {
			//in case of out of boundaries assign closest value
			if (x < d_x[i]) { d_interp_y[idx] = d_y[i]; break; }
			if (x > d_x[J - 1]) { d_interp_y[idx] = d_y[J - 1]; break; }
			if (x >= d_x[i] && x <= d_x[i + 1]) {
				double x0 = d_x[i];
				double x1 = d_x[i + 1];
				double y0 = d_y[i];
				double y1 = d_y[i + 1];

				// Perform linear interpolation
				d_interp_y[idx] = y0 + (x - x0) / (x1 - x0) * (y1 - y0);
				break;
			}
		}
	}
}


//Give a randState to each CUDA thread from which it can sample from
__global__ void init_rng(unsigned int seed, hiprandState* state)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(seed, idx, 0, &state[idx]);
}

__global__ void gen_x(hiprandState* state, double* x)
{
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	hiprandState localState = state[idx];
	x[idx] = hiprand_uniform_double(&localState);
	state[idx] = localState;
}





double CGMY_CUDA1(double S0, double  K, double  T, double  r, double  C, double  G, double  M, double  Y, double q) {

	////////////////////////////////CDF COMPUTATION////////////////////////////////

	// Number of points to generate and CUDA parameters
	const int N_sim = 1e6;
	const int threadsPerBlock = 1024;
	const int numBlocks = (N_sim + threadsPerBlock - 1) / threadsPerBlock;

	//COS parameters
	double L = 8;
	const int N_COS_TERMS = int(pow(2, 14));
	double a = -L * sqrt(T);
	double b = L * sqrt(T);

	//size of a pdf/cdf
	int J = 1000;
	double x_min = -4;
	double x_max = 4;
	double dx = (x_max - x_min) / J;


	double* h_x = new double[J];
	for (int j = 0; j < J; j++) h_x[j] = x_min + j * dx;

	double* f_x = new double[J];

	COS_CGMY_pdf(h_x, f_x, J, N_COS_TERMS, T, S0, r, q, C, G, M, Y, a, b);

	int J_size = 0;
	double F_j = f_x[0];
	for (int j = 0; j < J; j++)
	{
		F_j += (j == 0) ? 0 : f_x[j] * dx;
		if (0.05 <= F_j && F_j <= 0.95) J_size += 1;
	}

	double* F_x;
	hipMallocManaged((void**)&F_x, J_size * sizeof(double));
	double* x;
	hipMallocManaged((void**)&x, J_size * sizeof(double));

	F_j = f_x[0]; int k = 0;
	for (int j = 0; j < J; j++)
	{
		F_j += (j == 0) ? 0 : f_x[j] * dx;
		if (0.05 <= F_j && F_j <= 0.95)
		{
			F_x[k] = F_j;
			x[k] = h_x[j];
			k++;
		}
		//cout << f_x[j] <<"\t" << h_x[j] << endl;
	}



	delete[] f_x;

	double w = -C * tgamma(-Y) * (pow(M - 1, Y) - pow(M, Y) + pow(G + 1, Y) - pow(G, Y));
	double mu = r + w - q;


	////////////////////////////////SAMPLING OF UNIFORMAL VARIABLES////////////////////////////////
	auto start = high_resolution_clock::now();
	auto stop = high_resolution_clock::now();
	auto duration = duration_cast<microseconds>(stop - start);


	double* d_uniform;
	//Allocate Unified Memory � accessible from CPU or GPU
	hipMalloc((void**)&d_uniform, N_sim * sizeof(double));

	// Create a device pointer on the host, to hold the random states
	hiprandState* d_state;
	hipMalloc((void**)&d_state, N_sim * sizeof(hiprandState));

	unsigned long seed = time(NULL);
	// Init the random states
	init_rng << <numBlocks, threadsPerBlock >> > (seed, d_state);
	// Generate numbers in GPU
	gen_x << <numBlocks, threadsPerBlock >> > (d_state, d_uniform);

	////////////////////////////////INTERPOLATING WITH CDF////////////////////////////////

	double* d_ST;
	hipMallocManaged((void**)&d_ST, N_sim * sizeof(double));

	linearInterpolateKernel << <numBlocks, threadsPerBlock >> > (F_x, x, d_uniform, d_ST, J_size, N_sim);

	hipFree(d_uniform);
	hipFree(F_x);
	hipFree(x);

	////////////////////////////////PRICING OPTIONS////////////////////////////////

	float h_price;
	float* d_price;

	hipMalloc((void**)&d_price, sizeof(float));
	hipMemset(d_price, 0, sizeof(float));

	PriceByMC << <numBlocks, threadsPerBlock >> > (d_price, S0, r, q, w, T, K, N_sim, d_ST);

	hipMemcpy(&h_price, d_price, sizeof(float), hipMemcpyDeviceToHost);
	stop = high_resolution_clock::now(); duration = duration_cast<microseconds>(stop - start);
	hipFree(d_price);

	return double(h_price);
}

